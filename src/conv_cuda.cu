
#include <hip/hip_runtime.h>
#if (__CUDA_ARCH__ >= 530)
#define __CUDA_NO_HALF2_OPERATORS__ // block half2 vector math operators
#include <cuda_fp16.h> // define half/half2 types, without half2 operators
#endif

// real/complex conjugation
inline __host__ __device__ float conj(const float a) {
    return a; 
}
inline __host__ __device__ double conj(const double a) {
    return a;
}
#if (__CUDA_ARCH__ >= 530)
inline __host__ __device__ half conj(const half a) {
    return a;
}
#endif

inline __host__ __device__ float2 conj(const float2 a) {
    return make_float2(a.x, -a.y); 
}
inline __host__ __device__ double2 conj(const double2 a) {
    return make_double2(a.x, -a.y); 
}
#if (__CUDA_ARCH__ >= 530)
inline __host__ __device__ half2 conj(const half2 a) {
    return make_half2(a.x, -a.y); 
}
#endif


// complex multiplication
inline __host__ __device__ float2 operator*(const float2 a, const float2 b) {
    return make_float2(a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x);
}
inline __host__ __device__ double2 operator*(const double2 a, const double2 b) {
    return make_double2(a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x);
}
#if (__CUDA_ARCH__ >= 530)
inline __host__ __device__ half2 operator*(const half2 a, const half2 b) {
    return make_half2(a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x);
}
#endif

inline __host__ __device__ float2 operator*(const float2 a, const float b){
    return make_float2(b*a.x, b*a.y);
}
inline __host__ __device__ double2 operator*(const double2 a, const double b){
    return make_double2(b*a.x, b*a.y);
}
#if (__CUDA_ARCH__ >= 530)
inline __host__ __device__ half2 operator*(const half2 a, const half b){
    return make_half2(b*a.x, b*a.y);
}
#endif

// real/complex addition/assignment
inline __host__ __device__ void operator+=(float2 &a, const float2 b){
    a.x += b.x;
    a.y += b.y;
}
inline __host__ __device__ void operator+=(double2 &a, const double2 b){
    a.x += b.x;
    a.y += b.y;
}
#if (__CUDA_ARCH__ >= 530)
inline __host__ __device__ void operator+=(half2 &a, const half2 b){
    a.x += b.x;
    a.y += b.y;
}
#endif


/*
* Compute the cross correlation of two sets of data. The data will be
* correlated in the first dimension. M >= N must be satisfied.
*
* Inputs:
*  x:         first signal  (M x S)
*  y:         second signal (N x S)
*
* Outputs:
*  z:         resulting cross correlation
*
*
*/
# ifndef M
__constant__ size_t M; 
# endif
# ifndef N
__constant__ size_t N;
# endif
# ifndef L
__constant__ size_t L; // total number of lags
# endif
# ifndef L0
__constant__ int L0; // starting lag
# endif

// xcorr template
template <typename T>
inline __device__ void conv_temp(const T* x, const T* y, T* z, T za){
    /*    xcorr_temp(const T* x, const T* y, T* z, T za)
     x, y: input array pointer(s)
     za: 0 value for the data type
     z:    output array pointer
     cross correlation 
    */

    // get lag and stride indices
    const int    l = threadIdx.x + blockDim.x*blockIdx.x; // lag index 
    const size_t s = threadIdx.y + blockDim.y*blockIdx.y; // slice
    
    // if valid lag indices, multiply and accumulate in-place
    if(l < L)
        # pragma unroll
        for(int i = 0, j = L0 - l; i < M || j < N; ++i, ++j)
            if(0 <= i && i < M && 0 <= j && j < N) // signal in bounds
                za += x[i+s*M] * conj(y[j+s*N]); // accum the cross product

    // output result:
    if(l < L)
        z[l+s*L] = za;
}

// xcorr kernels
__global__ void convf(const float* x, const float* y, float* z){
    conv_temp<float>(x, y, z, 0.0f);
}

__global__ void conv(const double* x, const double* y, double* z){
    conv_temp<double>(x, y, z, 0.0);
}
#if (__CUDA_ARCH__ >= 530)
__global__ void convh(const ushort* x, const ushort* y, ushort* z){
    conv_temp<half>((half*)x, (half*)y, (half*)z, 0.0f);
}
#endif
__global__ void convcf(const float2* x, const float2* y, float2* z){
    conv_temp<float2>(x, y, z, make_float2(0.0f,0.0f));
}

__global__ void convc(const double2* x, const double2* y, double2* z){
    conv_temp<double2>(x, y, z, make_double2(0.0,0.0));
}
#if (__CUDA_ARCH__ >= 530)
__global__ void convch(const ushort2* x, const ushort2* y, ushort2* z){
    conv_temp<half2>((half2*)x, (half2*)y, (half2*)z, make_half2(0.0f, 0.0f));
}
#endif


