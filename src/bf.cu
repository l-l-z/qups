#include "hip/hip_runtime.h"
// # include "hip/hip_vector_types.h" // vector math

// # include "sizes.cu" // size defines

# include "interpd.cu" // samplers using constant sizing

// # include "half2_math.h" // vector math for half types only 


/* Creates the positions for a linear array aperture given it's description
*
* All positions are in projetive coordinates.
*
* Must be run with a kernel size equal to the number of elements in the 
* array.
*
* Inputs:
*   Pn:     Vector of positions
*   Pn0:    Initial position
*   dPn:    Interelement difference
*
*
*/ 

__global__ void pos_step_rng_lenf(float3 * Pn, const float3 Pn0, const float3 dPn){
    const uint idx = threadIdx.x + blockIdx.x * blockDim.x;
    Pn[idx] = Pn0 + idx * dPn;
}

__global__ void pos_step_rng_len(double3 * Pn, const double3 Pn0, const double3 dPn){
    const uint idx = threadIdx.x + blockIdx.x * blockDim.x;
    Pn[idx] = Pn0 + idx * dPn;
}


/*
* Delay and sum the given data at the given pixels
*
* Given a set of pixels, (virtual or plane wave) transmitter locations, 
* receiver locations, as well as a datacube equipped with a time, 
* transmitter and receiver axis, perform simple delay-and-sum beamforming. 
* The data is linearly interpolated at the sample time. An image is 
* generated for each receiver element. Summation across the transmitters 
* and receivers is implicit.
*
* All positions are in vector coordinates. 
* 
* If the virtual transmitter 
* normal has a fourth component that is 0, this indicates that the 
* transmission should be treated as a plane wave transmission instead of a 
* virtual source transmission. 
* 
* The value of t = 0 must be the time when the peak of the wavefront 
* reaches the virtual source location. Because this time must be the same 
* for all transmits, the datacube must be stitched together in such a way 
* that for all transmits, the same time axis is used
*
* Inputs:
*  y:           complex pixel values per channel (I)
*  Pi:          pixel positions (3 x I)
*  Pr:          receiver positions (3 x N)
*  Pv:          (virtual) transmitter positions (3 x M)
*  Nv:          (virtual) transmitter normal (3 x M)
*  x:           datacube of complex sample values (T x M x N)
*  t0:          initial time for the data
*  fs:          sampling frequency of the data
*  cinv:        inverse of the speed of sound used for beamforming
* 
* I -> pixels, M -> transmitters, N -> receivers, T -> time samples
*
*/

__global__ void DASf(float2 * __restrict__ y, 
    const float * __restrict__ Pi, const float * __restrict__ Pr, 
    const float * __restrict__ Pv, const float * __restrict__ Nv, 
	const float2 * __restrict__ a, const size_t * astride, 
    const float2 * __restrict__ x, const int iflag,
	const float t0, const float fs, const float cinv) {

    // get starting index of this pixel
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    // get image coordinates
    const size_t I1 = QUPS_I1, I2 = QUPS_I2, I3 = QUPS_I3; // rename for readability
    const size_t i1 = (tid             % I1); // index in I1
    const size_t i2 = (tid /  I1     ) % I2 ; // index in I2
    const size_t i3 = (tid / (I1 * I2) % I3); // index in I3
    const size_t abase = i1 * astride[0] + i2 * astride[1] + i3 * astride[2]; // base index for this pixel

    // reinterpret inputs as vector pointers (makes loading faster and indexing easier)
    const float3 * pi = reinterpret_cast<const float3*>(Pi); // 3 x I
    const float3 * pr = reinterpret_cast<const float3*>(Pr); // 3 x N
    const float3 * pv = reinterpret_cast<const float3*>(Pv); // 3 x M
    const float3 * nv = reinterpret_cast<const float3*>(Nv); // 3 x M
    
    // rename for readability
    const size_t N = QUPS_N, M = QUPS_M, T = QUPS_T, I = QUPS_I;
            
    // temp vars
    const float2 zero_v = make_float2(0.0f);
    float2 w;
    float rf, dv, dr, tau;
    float2 val, pix = zero_v;
    float3 rv;

    // if valid pixel, for each tx/rx
    if(tid < I){
        # pragma unroll
        for(size_t m = 0; m < M; ++m){
            # pragma unroll
            for(size_t n = 0; n < N; ++n){
                // 2-way virtual path distance
                rv = pi[tid] - pv[m]; // (virtual) transmit to pixel vector 
                
                dv = QUPS_VS ? // tx path length
                    copysign(length(rv), dot(rv, nv[m])) // virtual source
                    : dot(rv, nv[m]); // plane wave
                
                dr = length(pi[tid] - pr[n]); // rx path length

                // data/time index number
                tau = (cinv * (dv + dr) - t0);
                w = make_float2(1.0f, 0.0f); // TODO: enable demod: make_float2(cospi(2*fs/4*tau), -sinpi(2*fs/4*tau));
                rf =  tau * fs;

                // sample the trace
                val = samplef(&x[(n + m * N) * T], rf, iflag, zero_v); // out of bounds: extrap 0

                // accumulate tx here: add to pixel value
                pix += val * w * a[abase + n * astride[3] + m * astride[4]];
            }
        }
        y[tid] = pix; // output value 
    }
}

__global__ void DAS(double2 * __restrict__ y, 
    const double * __restrict__ Pi, const double * __restrict__ Pr, 
    const double * __restrict__ Pv, const double * __restrict__ Nv, 
    const double2 * __restrict__ a, const size_t * astride,
	const double2 * __restrict__ x, const int iflag,
	const double t0, const double fs, const double cinv) {

    // get starting index of this pixel
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    // get image coordinates
    const size_t I1 = QUPS_I1, I2 = QUPS_I2, I3 = QUPS_I3; // rename for readability
    const size_t i1 = (tid             % I1); // index in I1
    const size_t i2 = (tid /  I1     ) % I2 ; // index in I2
    const size_t i3 = (tid / (I1 * I2) % I3); // index in I3
    const size_t abase = i1 * astride[0] + i2 * astride[1] + i3 * astride[2]; // base index for this pixel

    // reinterpret inputs as vector pointers (makes loading faster and indexing easier)
    const double3 * pi = reinterpret_cast<const double3*>(Pi); // 4 x I
    const double3 * pr = reinterpret_cast<const double3*>(Pr); // 3 x N
    const double3 * pv = reinterpret_cast<const double3*>(Pv); // 3 x M
    const double3 * nv = reinterpret_cast<const double3*>(Nv); // 3 x M
    
    // rename for readability
    const size_t N = QUPS_N, M = QUPS_M, T = QUPS_T, I = QUPS_I;
            
    // temp vars
    const double2 zero_v = make_double2(0.0);
    double2 w;
    double rf, dv, dr, tau;
    double2 val, pix = zero_v;
    double3 rv;

    // if valid pixel, for each tx/rx
    if(tid < I){
        # pragma unroll
        for(size_t m = 0; m < M; ++m){
            # pragma unroll
            for(size_t n = 0; n < N; ++n){
                // 2-way virtual path distance
                rv = pi[tid] - pv[m]; // (virtual) transmit to pixel vector 
                
                dv = QUPS_VS ? // tx path length
                    copysign(length(rv), dot(rv, nv[m])) // virtual source
                    : dot(rv, nv[m]); // plane wave
                
                dr = length(pi[tid] - pr[n]); // rx path length

                // data/time index number
                tau = (cinv * (dv + dr) - t0);
                w = make_double2(1.0,0.0); // TODO: enable demod: make_double2(cospi(2*fs/4*tau), -sinpi(2*fs/4*tau));
                rf =  tau * fs;

                // sample the trace
                val = sample(&x[(n + m * N) * T], rf, iflag, zero_v); // out of bounds: extrap 0

                // accumulate tx here: add to pixel value
                pix += val * w * a[abase + n * astride[3] + m * astride[4]];
            }            
        }
        y[tid] = pix; // output value
    }
}

__global__ void DASh(ushort2 * __restrict__ y, 
    const float * __restrict__ Pi, const float * __restrict__ Pr, 
    const float * __restrict__ Pv, const float * __restrict__ Nv, 
	const ushort2 * __restrict__ a, const size_t * astride, 
    const ushort2 * __restrict__ x, const int iflag,
	const float t0, const float fs, const float cinv) {

    // get starting index of this pixel
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    // get image coordinates
    const size_t I1 = QUPS_I1, I2 = QUPS_I2, I3 = QUPS_I3; // rename for readability
    const size_t i1 = (tid             % I1); // index in I1
    const size_t i2 = (tid /  I1     ) % I2 ; // index in I2
    const size_t i3 = (tid / (I1 * I2) % I3); // index in I3
    const size_t abase = i1 * astride[0] + i2 * astride[1] + i3 * astride[2]; // base index for this pixel

    // reinterpret inputs as vector pointers (makes loading faster and indexing easier)
    const float3 * pi = reinterpret_cast<const float3*>(Pi); // 3 x I
    const float3 * pr = reinterpret_cast<const float3*>(Pr); // 3 x N
    const float3 * pv = reinterpret_cast<const float3*>(Pv); // 3 x M
    const float3 * nv = reinterpret_cast<const float3*>(Nv); // 3 x M

    
    // rename for readability
    const size_t N = QUPS_N, M = QUPS_M, T = QUPS_T, I = QUPS_I;
            
    // temp vars
    const half2 zero_v = make_half2(0.f, 0.f);
    half2 w;
    float rf, dv, dr, tau;
    half2 val, pix = zero_v;
    float3 rv;

    // if valid pixel, for each tx/rx
    if(tid < I){
        # pragma unroll
        for(size_t m = 0; m < M; ++m){
            # pragma unroll
            for(size_t n = 0; n < N; ++n){
                // 2-way virtual path distance
                rv = pi[tid] - pv[m]; // (virtual) transmit to pixel vector 
                
                dv = QUPS_VS ? // tx path length
                    copysign(length(rv), dot(rv, nv[m])) // virtual source
                    : dot(rv, nv[m]); // plane wave
                
                dr = length(pi[tid] - pr[n]); // rx path length

                // data/time index number
                tau = (cinv * (dv + dr) - t0);
                w = make_half2(1.0f, 0.0f); // TODO: enable demod: make_half2(cospi(2*fs/4*tau), -sinpi(2*fs/4*tau));
                rf =  tau * fs;

                // sample the trace
                val = sampleh(&x[(n + m * N) * T], rf, iflag, zero_v); // out of bounds: extrap 0

                // accumulate tx here: add to pixel value
                pix += val * w * u2h(a[abase + n * astride[3] + m * astride[4]]);
            }
        }
        y[tid] = h2u(pix); // output value 
    }
}



/*
* Delay the given data at the given pixels and sum over transmits
*
* Given a set of pixels, (virtual or plane wave) transmitter locations, 
* receiver locations, as well as a datacube equipped with a time, 
* transmitter and receiver axis, perform simple delay-and-sum beamforming. 
* The data is linearly interpolated at the sample time. An image is 
* generated for each receiver element. 
*
* All positions are in vector coordinates. 
* 
* If the virtual transmitter 
* normal has a fourth component that is 0, this indicates that the 
* transmission should be treated as a plane wave transmission instead of a 
* virtual source transmission. 
* 
* The value of t = 0 must be the time when the peak of the wavefront 
* reaches the virtual source location. Because this time must be the same 
* for all transmits, the datacube must be stitched together in such a way 
* that for all transmits, the same time axis is used
*
* Inputs:
*  y:           complex pixel values per channel (N x I)
*  Pi:          pixel positions (3 x I)
*  Pr:          receiver positions (3 x N)
*  Pv:          (virtual) transmitter positions (3 x M)
*  Nv:          (virtual) transmitter normal (3 x M)
*  x:           datacube of complex sample values (T x M x N)
*  t0:          initial time for the data
*  fs:          sampling frequency of the data
*  cinv:        inverse of the speed of sound used for beamforming
* 
* I -> pixels, M -> transmitters, N -> receivers, T -> time samples
*
*/

__global__ void SYNf(float2 * __restrict__ y, 
    const float * __restrict__ Pi, const float * __restrict__ Pr, 
    const float * __restrict__ Pv, const float * __restrict__ Nv, 
    const float2 * __restrict__ a, const size_t * astride,
    const float2 * __restrict__ x, const int iflag,
	const float t0, const float fs, const float cinv) {

    // get starting index of this pixel
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    // get image coordinates (Isz)
    const size_t I1 = QUPS_I1, I2 = QUPS_I2, I3 = QUPS_I3; // rename for readability
    const size_t i1 = (tid             % I1); // index in I1
    const size_t i2 = (tid /  I1     ) % I2 ; // index in I2
    const size_t i3 = (tid / (I1 * I2) % I3); // index in I3
    const size_t abase = i1 * astride[0] + i2 * astride[1] + i3 * astride[2]; // base index for this pixel

    // reinterpret inputs as vector pointers (makes loading faster and indexing easier)
    const float3 * pi = reinterpret_cast<const float3*>(Pi); // 4 x I
    const float3 * pr = reinterpret_cast<const float3*>(Pr); // 3 x N
    const float3 * pv = reinterpret_cast<const float3*>(Pv); // 3 x M
    const float3 * nv = reinterpret_cast<const float3*>(Nv); // 3 x M
    
    // rename for readability
    const size_t N = QUPS_N, M = QUPS_M, T = QUPS_T, I = QUPS_I;
            
    // temp vars
    const float2 zero_v = make_float2(0.0f);
    float rf, dv, dr;
    float2 val;
    float3 rv;

    // if valid pixel, for each tx/rx
    if(tid < I){
        # pragma unroll
        for(size_t m = 0; m < M; ++m){
            # pragma unroll
            for(size_t n = 0; n < N; ++n){
                // 2-way virtual path distance
                rv = pi[tid] - pv[m]; // (virtual) transmit to pixel vector 
                
                dv = QUPS_VS ? // tx path length
                    copysign(length(rv), dot(rv, nv[m])) // virtual source
                    : dot(rv, nv[m]); // plane wave
                
                dr = length(pi[tid] - pr[n]); // rx path length

                // data/time index number
                rf = (cinv * (dv + dr) - t0) * fs;

                // sample the trace
                val = samplef(&x[(n + m * N) * T], rf, iflag, zero_v); // out of bounds: extrap 0

                // apply apodization
                val *= a[abase + n * astride[3] + m * astride[4]]; // index as I x N

                // accumulate tx here: add to pixel value
                y[tid + n*I] += val;
            }
        }
    }
}

__global__ void SYN(double2 * __restrict__ y, 
    const double * __restrict__ Pi, const double * __restrict__ Pr, 
    const double * __restrict__ Pv, const double * __restrict__ Nv, 
	const double2 * __restrict__ a, const size_t * astride,
	const double2 * __restrict__ x, const int iflag,
	const double t0, const double fs, const double cinv) {

    // get starting index of this pixel
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    // get image coordinates
    const size_t I1 = QUPS_I1, I2 = QUPS_I2, I3 = QUPS_I3; // rename for readability
    const size_t i1 = (tid             % I1); // index in I1
    const size_t i2 = (tid /  I1     ) % I2 ; // index in I2
    const size_t i3 = (tid / (I1 * I2) % I3); // index in I3
    const size_t abase = i1 * astride[0] + i2 * astride[1] + i3 * astride[2]; // base index for this pixel

    // reinterpret inputs as vector pointers (makes loading faster and indexing easier)
    const double3 * pi = reinterpret_cast<const double3*>(Pi); // 3 x I
    const double3 * pr = reinterpret_cast<const double3*>(Pr); // 3 x N
    const double3 * pv = reinterpret_cast<const double3*>(Pv); // 3 x M
    const double3 * nv = reinterpret_cast<const double3*>(Nv); // 3 x M
    
    // rename for readability
    const size_t N = QUPS_N, M = QUPS_M, T = QUPS_T, I = QUPS_I;
            
    // temp vars
    const double2 zero_v = make_double2(0.0);
    double rf, dv, dr;
    double2 val;
    double3 rv;

    // if valid pixel, for each tx/rx
    if(tid < I){
        # pragma unroll
        for(size_t m = 0; m < M; ++m){
            # pragma unroll
            for(size_t n = 0; n < N; ++n){
                // 2-way virtual path distance
                rv = pi[tid] - pv[m]; // (virtual) transmit to pixel vector 
                
                dv = QUPS_VS ? // tx path length
                    copysign(length(rv), dot(rv, nv[m])) // virtual source
                    : dot(rv, nv[m]); // plane wave
                
                dr = length(pi[tid] - pr[n]); // rx path length

                // data/time index number
                rf = (cinv * (dv + dr) - t0) * fs;

                // sample the trace
                val = sample(&x[(n + m * N) * T], rf, iflag, zero_v); // out of bounds: extrap 0

                // accumulate tx here: add to pixel value
                y[tid + n*I] += val * a[abase + n * astride[3] + m * astride[4]]; // index as I x N
            }
        }
    }
}


/*
* Beamform the data at the given pixels.
*
* Given a set of pixels, (virtual or plane wave) transmitter locations, 
* receiver locations, as well as a datacube equipped with a time, 
* transmitter and receiver axis, beamforming the data without summation. 
* The data is linearly interpolated at the sample time. 
*
* All positions are in vector coordinates. 
* 
* If the virtual transmitter normal has a fourth component that is 0, this 
* indicates that the transmission should be treated as a plane wave 
* transmission instead of a virtual source (focused) transmission. 
* 
* The value of t = 0 must be the time when the peak of the wavefront 
* reaches the virtual source location. Because this time must be the same 
* for all transmits, the datacube must be stitched together in such a way 
* that for all transmits, the same time axis is used.
*
* Inputs:
*  y:           complex pixel values per transmit/channel (M x N x I)
*  Pi:          pixel positions (3 x I)
*  Pr:          receiver positions (3 x N)
*  Pv:          (virtual) transmitter positions (3 x M)
*  Nv:          (virtual) transmitter normal (3 x M)
*  x:           datacube of complex sample values (T x M x N)
*  t0:          initial time for the data
*  fs:          sampling frequency of the data
*  cinv:        inverse of the speed of sound used for beamforming
* 
* I -> pixels, M -> transmitters, N -> receivers, T -> time samples
*
*/

__global__ void BFf(float2 * __restrict__ y, 
    const float * __restrict__ Pi, const float * __restrict__ Pr, 
    const float * __restrict__ Pv, const float * __restrict__ Nv, 
	const float2 * __restrict__ a, const size_t * astride,
	const float2 * __restrict__ x, const int iflag,
	const float t0, const float fs, const float cinv) {

    // get starting index of this pixel
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    // get image coordinates
    const size_t I1 = QUPS_I1, I2 = QUPS_I2, I3 = QUPS_I3; // rename for readability
    const size_t i1 = (tid             % I1); // index in I1
    const size_t i2 = (tid /  I1     ) % I2 ; // index in I2
    const size_t i3 = (tid / (I1 * I2) % I3); // index in I3
    const size_t abase = i1 * astride[0] + i2 * astride[1] + i3 * astride[2]; // base index for this pixel

    // reinterpret inputs as vector pointers (makes loading faster and indexing easier)
    const float3 * pi = reinterpret_cast<const float3*>(Pi); // 4 x I
    const float3 * pr = reinterpret_cast<const float3*>(Pr); // 3 x N
    const float3 * pv = reinterpret_cast<const float3*>(Pv); // 3 x M
    const float3 * nv = reinterpret_cast<const float3*>(Nv); // 3 x M
    
    // rename for readability
    const size_t N = QUPS_N, M = QUPS_M, T = QUPS_T, I = QUPS_I;
            
    // temp vars
    const float2 zero_v = make_float2(0.0f);
    float rf, dv, dr;
    float2 val;
    float3 rv;

    // if valid pixel, for each tx/rx
    if(tid < I){
        # pragma unroll
        for(size_t m = 0; m < M; ++m){
            # pragma unroll
            for(size_t n = 0; n < N; ++n){
                // 2-way virtual path distance
                rv = pi[tid] - pv[m]; // (virtual) transmit to pixel vector 
                
                dv = QUPS_VS ? // tx path length
                    copysign(length(rv), dot(rv, nv[m])) // virtual source
                    : dot(rv, nv[m]); // plane wave
                
                dr = length(pi[tid] - pr[n]); // rx path length

                // data/time index number
                rf = (cinv * (dv + dr) - t0) * fs;

                // sample the trace
                val = samplef(&x[(n + m * N) * T], rf, iflag, zero_v); // out of bounds: extrap 0

                // output value
                y[tid + n * I + m * I * N] = val * a[abase + n * astride[3] + m * astride[4]]; // index as I x N x M
            }            
        }
    }
}


__global__ void BF(double2 * __restrict__ y, 
    const double * __restrict__ Pi, const double * __restrict__ Pr, 
    const double * __restrict__ Pv, const double * __restrict__ Nv, 
	const double2 * __restrict__ a, const size_t * astride,
	const double2 * __restrict__ x, const int iflag,
	const double t0, const double fs, const double cinv) {

    // get starting index of this pixel
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    // get image coordinates
    const size_t I1 = QUPS_I1, I2 = QUPS_I2, I3 = QUPS_I3; // rename for readability
    const size_t i1 = (tid             % I1); // index in I1
    const size_t i2 = (tid /  I1     ) % I2 ; // index in I2
    const size_t i3 = (tid / (I1 * I2) % I3); // index in I3
    const size_t abase = i1 * astride[0] + i2 * astride[1] + i3 * astride[2]; // base index for this pixel

    // reinterpret inputs as vector pointers (makes loading faster and indexing easier)
    const double3 * pi = reinterpret_cast<const double3*>(Pi); // 4 x I
    const double3 * pr = reinterpret_cast<const double3*>(Pr); // 3 x N
    const double3 * pv = reinterpret_cast<const double3*>(Pv); // 3 x M
    const double3 * nv = reinterpret_cast<const double3*>(Nv); // 3 x M
    
    // rename for readability
    const size_t N = QUPS_N, M = QUPS_M, T = QUPS_T, I = QUPS_I;
            
    // temp vars
    const double2 zero_v = make_double2(0.0f);
    double rf, dv, dr;
    double2 val;
    double3 rv;

    // if valid pixel, for each tx/rx
    if(tid < I){
        # pragma unroll
        for(size_t m = 0; m < M; ++m){
            # pragma unroll
            for(size_t n = 0; n < N; ++n){
                // 2-way virtual path distance
                rv = pi[tid] - pv[m]; // (virtual) transmit to pixel vector 
                
                dv = QUPS_VS ? // tx path length
                    copysign(length(rv), dot(rv, nv[m])) // virtual source
                    : dot(rv, nv[m]); // plane wave
                
                dr = length(pi[tid] - pr[n]); // rx path length

                // data/time index number
                rf = (cinv * (dv + dr) - t0) * fs;

                // sample the trace
                val = sample(&x[(n + m * N) * T], rf, iflag, zero_v); // out of bounds: extrap 0

                // output value
                y[tid + n * I + m * I * N] = val * a[abase + n * astride[3] + m * astride[4]]; // index as I x N x M
            }            
        }
    }
}


/*
* Beamforming delays at the given pixels.
*
* Given a set of pixels, (virtual or plane wave) transmitter locations, 
* receiver locations, as well as a datacube equipped with a time, 
* transmitter and receiver axis, compute the sample times corresponding to 
* when the peak of the response from an ideal scatterer arrives at the 
* receiver. 
*
* All positions are in vector coordinates. 
* 
* If the virtual transmitter normal has a fourth component that is 0, this 
* indicates that the transmission should be treated as a plane wave 
* transmission instead of a virtual source (focused) transmission. 
* 
* The value of t = 0 must be the time when the peak of the wavefront 
* reaches the virtual source location. Because this time must be the same 
* for all transmits, the datacube must be stitched together in such a way 
* that for all transmits, the same time axis is used.
*
* Inputs:
*  tau:         sample time per transmit/channel/pixel (M x N x I)
*  Pi:          pixel positions (3 x I)
*  Pr:          receiver positions (3 x N)
*  Pv:          (virtual) transmitter positions (3 x M)
*  Nv:          (virtual) transmitter normal (3 x M)
*  t0:          initial time for the data
*  fs:          sampling frequency of the data
*  cinv:        inverse of the speed of sound used for beamforming
* 
* I -> pixels, M -> transmitters, N -> receivers, T -> time samples
*
*/

__global__ void delaysf(float * __restrict__ tau, 
    const float * __restrict__ Pi, const float * __restrict__ Pr, 
    const float * __restrict__ Pv, const float * __restrict__ Nv, 
	const float cinv) {

    // get starting index of this pixel
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    // reinterpret inputs as vector pointers (makes loading faster and indexing easier)
    const float3 * pi = reinterpret_cast<const float3*>(Pi); // 3 x I
    const float3 * pr = reinterpret_cast<const float3*>(Pr); // 3 x N
    const float3 * pv = reinterpret_cast<const float3*>(Pv); // 3 x M
    const float3 * nv = reinterpret_cast<const float3*>(Nv); // 3 x M
    
    // rename for readability
    const size_t N = QUPS_N, M = QUPS_M, I = QUPS_I;

    // temp vars
    float dv, dr;
    float3 rv;
    
    // if valid pixel, for each tx/rx
    if(tid < I){
        # pragma unroll
        for(size_t m = 0; m < M; ++m){
            # pragma unroll
            for(size_t n = 0; n < N; ++n){
                // 2-way virtual path distance
                rv = pi[tid] - pv[m]; // (virtual) transmit to pixel vector 
                
                dv = QUPS_VS ? // tx path length
                    copysign(length(rv), dot(rv, nv[m])) // virtual source
                    : dot(rv, nv[m]); // plane wave
                
                dr = length(pi[tid] - pr[n]); // rx path length

                // output time
                tau[tid + n * I + m * I * N] = cinv * (dv + dr);
            }            
        }
    }
}


__global__ void delays(double * __restrict__ tau, 
    const double * __restrict__ Pi, const double * __restrict__ Pr, 
    const double * __restrict__ Pv, const double * __restrict__ Nv, 
	const double cinv) {

    // get starting index of this pixel
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    // reinterpret inputs as vector pointers (makes loading faster and indexing easier)
    const double3 * pi = reinterpret_cast<const double3*>(Pi); // 3 x I
    const double3 * pr = reinterpret_cast<const double3*>(Pr); // 3 x N
    const double3 * pv = reinterpret_cast<const double3*>(Pv); // 3 x M
    const double3 * nv = reinterpret_cast<const double3*>(Nv); // 3 x M
    
    // rename for readability
    const size_t N = QUPS_N, M = QUPS_M, I = QUPS_I;

    // temp vars
    double dv, dr;
    double3 rv;
    
    // if valid pixel, for each tx/rx
    if(tid < I){
        # pragma unroll
        for(size_t m = 0; m < M; ++m){
            # pragma unroll
            for(size_t n = 0; n < N; ++n){
                // 2-way virtual path distance
                rv = pi[tid] - pv[m]; // (virtual) transmit to pixel vector 
                
                dv = QUPS_VS ? // tx path length
                    copysign(length(rv), dot(rv, nv[m])) // virtual source
                    : dot(rv, nv[m]); // plane wave
                
                dr = length(pi[tid] - pr[n]); // rx path length

                // output time
                tau[tid + n * I + m * I * N] = cinv * (dv + dr);
            }
        }
    }
}
